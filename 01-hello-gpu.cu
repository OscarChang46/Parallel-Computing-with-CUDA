
#include <hip/hip_runtime.h>
#include <stdio.h>

void helloCPU()
{
  printf("Hello from the CPU.\n");
}

/*
 * Refactor the `helloGPU` definition to be a kernel
 * that can be launched on the GPU. Update its message
 * to read "Hello from the GPU!"
 */

__global__ void helloGPU()
{
  printf("Hello also from the GPU.\n");
}

int main()
{
    helloGPU<<<1,1>>>();
    hipDeviceSynchronize();
    helloCPU();
    helloGPU<<<1,1>>>();
    hipDeviceSynchronize();
    
    

  /*
   * Add code below to synchronize on the completion of the
   * `helloGPU` kernel completion before continuing the CPU
   * thread.
   */
  

  /*
   * Refactor this call to `helloGPU` so that it launches
   * as a kernel on the GPU.
   */

  
}
